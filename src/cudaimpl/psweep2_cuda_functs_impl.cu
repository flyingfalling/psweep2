#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <psweep2_cuda_functs_impl.h>

void real_set_cuda_device( const size_t& idx )
{
  checkCudaErrors( hipSetDevice(idx) );
}

std::vector<size_t> findlegaldevs_byname(const std::string& devname)
{
  std::vector<size_t> ret;
  
  int deviceCount = 0;
 hipError_t error_id = hipGetDeviceCount(&deviceCount) ;
  if (error_id != hipSuccess)
    {
      fprintf(stderr, "hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
      fprintf(stderr, "Result = FAIL\n");
      exit(EXIT_FAILURE);
    }
  
  for(int dev = 0; dev < deviceCount; ++dev)
    {
      checkCudaErrors( hipSetDevice(dev) );
      hipDeviceProp_t deviceProp;
      checkCudaErrors( hipGetDeviceProperties(&deviceProp, dev) );
      if( devname.compare( std::string(deviceProp.name) ) == 0 )
	{
	  ret.push_back( dev );
	}
    }

  return ret;
}