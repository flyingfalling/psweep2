
#include <unit_tests/cuda_prog.h>

#include <hip/hip_runtime.h>
#include <cstdlib>
#include <vector>
#include <string>
#include <cstdio>


std::vector<size_t> find_legaldevs()
{
  std::vector<size_t> ret;

  
  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);
  if (error_id != hipSuccess)
    {
      fprintf(stderr, "hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
      fprintf(stderr, "Result = FAIL\n");
      exit(EXIT_FAILURE);
    }
  
  //  int driverVersion = 0;
  //  int runtimeVersion = 0;

  for (int dev = 0; dev < deviceCount; ++dev)
    {
      hipSetDevice(dev);
      hipDeviceProp_t deviceProp;
      hipGetDeviceProperties(&deviceProp, dev);
//      printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);
      /*
	char msg[256];
        SPRINTF(msg, "  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
	(float)deviceProp.totalGlobalMem/1048576.0f, (unsigned long long) deviceProp.totalGlobalMem);
        printf("%s", msg);
	printf("  (%2d) Multiprocessors, (%3d) CUDA Cores/MP:     %d CUDA Cores\n",
	deviceProp.multiProcessorCount,
	_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
	_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);
      */
      if( std::string(deviceProp.name).compare( "Tesla K80" ) == 0 )
	{
	  ret.push_back( dev );
	}
    }
  //hipDeviceReset();
  return ret;
}
