#include "hip/hip_runtime.h"

#include <unit_tests/cuda_prog.h>

#include <hip/hip_runtime.h>
#include <cstdlib>
#include <vector>
#include <string>
#include <cstdio>

#include <commontypes.h>

#include <hip/hip_runtime_api.h>


std::vector<size_t> find_legaldevs()
{
  std::vector<size_t> ret;

  
  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);
  if (error_id != hipSuccess)
    {
      fprintf(stderr, "hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
      fprintf(stderr, "Result = FAIL\n");
      exit(EXIT_FAILURE);
    }
  
  //  int driverVersion = 0;
  //  int runtimeVersion = 0;

  for (int dev = 0; dev < deviceCount; ++dev)
    {
      hipSetDevice(dev);
      hipDeviceProp_t deviceProp;
      hipGetDeviceProperties(&deviceProp, dev);
//      printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);
      /*
	char msg[256];
        SPRINTF(msg, "  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
	(float)deviceProp.totalGlobalMem/1048576.0f, (unsigned long long) deviceProp.totalGlobalMem);
        printf("%s", msg);
	printf("  (%2d) Multiprocessors, (%3d) CUDA Cores/MP:     %d CUDA Cores\n",
	deviceProp.multiProcessorCount,
	_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
	_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);
      */
      if( std::string(deviceProp.name).compare( "Tesla K80" ) == 0 )
	{
	  ret.push_back( dev );
	}
    }
  //hipDeviceReset();
  return ret;
}

__global__ void compDist( float64_t *res, float64_t *a, float64_t *b, int sizen )
{
  // Get our global thread ID
  int id = (blockIdx.x*blockDim.x) + threadIdx.x;
  
  // Make sure we do not go out of bounds
  if (id < sizen)
    {
      res[id] = a[id] - b[id];
      res[id] = (res[id]*res[id]);
    }
  //else
  //   { do nothing } 

  //return;
}

std::vector<float64_t> gpucomp( std::vector<float64_t>& est, std::vector<float64_t>& actual, size_t& cudadevnum )
{
  checkCudaErrors(hipSetDevice(cudadevnum)); //check errors? rofl.
  
  if(est.size() != actual.size())
    {
      fprintf(stderr,"REV: ERROR in cuda gpucomp, actual != est size!\n"); exit(1);
    }

  std::vector<float64_t> result( est.size(), -666 );
  
  float64_t* d_estptr;
  float64_t* d_actualptr;
  float64_t* d_resultptr;
  
  //Run the appropriate kernel
  checkCudaErrors(hipMalloc(&d_estptr, est.size()*sizeof(est[0]) ));
  checkCudaErrors(hipMalloc(&d_actualptr, actual.size()*sizeof(actual[0]) ));
  checkCudaErrors(hipMalloc(&d_resultptr, result.size()*sizeof(result[0]) ));
  
  checkCudaErrors(hipMemcpy(d_estptr, est.data(), est.size()*sizeof(est[0]), 
	     hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_actualptr, actual.data(), actual.size()*sizeof(actual[0]), 
	     hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_resultptr, result.data(), result.size()*sizeof(result[0]), 
	     hipMemcpyHostToDevice));

  //RUN KERNEL
  int blockSize=0;
  int gridSize=0;
 
  // Number of threads in each thread block
  blockSize = 1024;
 
  // Number of thread blocks in grid
  gridSize = 1; //(int)ceil((float)est.size()/blockSize);
  
  // Execute the kernel
  compDist<<<gridSize, blockSize>>>(d_resultptr, d_estptr, d_actualptr, (int)result.size());

  //REV: Do I need to synch it or something?
  hipDeviceSynchronize() ;

  checkCudaErrors(hipMemcpy( result.data(), d_resultptr, result.size()*sizeof(result[0]), hipMemcpyDeviceToHost ));

  checkCudaErrors(hipFree( d_estptr ));
		  checkCudaErrors(hipFree( d_actualptr ));
  checkCudaErrors( hipFree( d_resultptr ) );

  getLastCudaError("REV: Kernel execution failed");
  
  return result;  
  
}